
#include <hip/hip_runtime.h>
#include <cstdint>

namespace bls12_377 {
__device__ __constant__ uint32_t inv_p = 0xffffffff;
__device__ __constant__ uint32_t inv_q = 0xffffffff;
} // namespace bls12_377

namespace bls12_381 {
    __device__ __constant__ uint32_t inv_p = 0xfffcfffd;
    __device__ __constant__ uint32_t inv_q = 0xffffffff;
} // namespace bls12_377